#include "hip/hip_runtime.h"
#include <stdio.h>
#include "imageGenerator.h"
#include "constants.h"
#include "float3.h"
#include "ray.h"
#include "hitable.h"


//TODO write some comments

//--Optional
//TODO make the antialiasing sampling random.
////TODO Fare in modo che sia possibile costruire scene dinamicamente e gli oggetti delle scene
//				che vengono costruite verranno salvati nella memoria statica

/**
 * The nvidia's GPUs have 64kb of memory that can be filled with constant variables.
 * Using this memory in the raytracer gives two advantages:
 * 1 - Sligly increase the performance because it is cached
 * 2 - Sligly increase the performance because those variables are allocated only once for
 * 		thousands of cores
 * 	3- Once again, since they are allocated once for all the cores a noticable amount of memory
 * 		remain free
 */
//These are used to represent the camera
__constant__ float3 lowerLeftCorner;
__constant__ float3 vertical;
__constant__ float3 origin;
__constant__ float3 horizontal;
__constant__ sphere spheres[OBJNUMBER];

//Number of spheres
const int sn = 2;

void gpuErrorCheck(int i = 0){
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		    printf("Error %d: %s\n", i, hipGetErrorString(err));
}

/**
 * Given a ray, create the blend of colors
 */
__device__ float3 color(ray &r, int sphereNumber, int* hittedMaterial){

	hitRecord rec, tempRec;
	bool hitted = false;
	double closest = MAXFLOAT;

	for(int i = 0; i < sphereNumber; i++) {
		//Check if the ray hit the object
		//If there is already an hitted object it controls also if the new hitted object is closer than the previous
		if (spheres[i].hit(r, 0.0, closest, tempRec)) {
			hitted = true;
			closest = tempRec.t;
			rec = tempRec;
		}
	}

	//If the ray hitted something
	if (hitted) {
		//Calculate the normal of the hitted objed
		float3 normal = (r.pointAtParam(rec.t) - rec.c) / rec.r;
		//Draw the normal
		*hittedMaterial = rec.objId;
		return 0.5 * make_float3(normal.x + 1.0, normal.y + 1.0, normal.z + 1.0);
	} else { //Draw the background
		float y = unitVector(r.direction()).y;
		float t = 0.5 * (y + 1.0);
		return (1.0 - t) * make_float3(1.0, 1.0, 1.0) + t * make_float3(0.5, 0.7, 1.0);
	}
}

__global__ void kernel(int* imageGpu, int sphereNumber){
	__shared__ int hittedMaterial[THRDSIZE];
	__shared__ bool anti;

	if (threadIdx.x == 0) anti = false;

	hittedMaterial[threadIdx.x] = 0;

	int offset = blockIdx.x * blockDim.x;
	int w = offset % WIDTH + threadIdx.x;
	int h = int(offset / WIDTH);

	//u and v are used to translate a pixel coordinate on the scene
	float u =  float(w) / float(WIDTH);
	float v = float(h) / float(HEIGHT);

	//generate a ray that start from the origin and pass trough the center of a given pixel
	ray r(origin, lowerLeftCorner + (u * horizontal) + (v * vertical));
	//calculate the color that that ray sees
	float3 col = color(r, sphereNumber, &hittedMaterial[threadIdx.x]);

	__syncthreads();
	if (threadIdx.x == 0) {
		int firstHit = hittedMaterial[0];
		for (int i = 1; i < THRDSIZE; i++) {
			//printf("block %d , thread %d hits %d\n", blockIdx.x, threadIdx.x, hittedMaterial[0]);
			if(firstHit != hittedMaterial[i]) {
				anti = true;
			}
		}
	}
	__syncthreads();


	if (anti) { //Exec the antialiasing if necessary
		//Generate 9 ray equally spaced for each pixel
		for (int i = 0; i < 3; i ++) {
			for (int j = 0; j < 3; j ++){
				u =  (float(w) + 0.3 * i) / float(WIDTH);
				v = (float(h) + 0.3 * j) / float(HEIGHT);
				r = ray(origin, lowerLeftCorner + (u * horizontal) + (v * vertical));
				col = col + color(r, sphereNumber, &hittedMaterial[threadIdx.x]);
				col = col / 2;
			}
		}
		//in the end the value of color is the average color of the 9 rays
	}

	//Put the color seen by the ray in the memory address that correspond to the pixel
	imageGpu[addressConverter(h, w, 0)] = int(255.99 * col.x);
	imageGpu[addressConverter(h, w, 1)] = int(255.99 * col.y);
	imageGpu[addressConverter(h, w, 2)] = int(255.99 * col.z);

}

int main ()
{

	//Allocate a tridimensional vector that contains the image's data
	int *image;

	//hipHostAlloc is used to allocate paged memory on the host device, in this way the position of the memory will
	//never change. It is necessary if we want to use asynchronous loading of the data in the gpu
	hipHostAlloc((void**)&image, WIDTH * HEIGHT * BYTESPERPIXEL * sizeof( int ), hipHostMallocDefault);
	gpuErrorCheck();

	memset((int*) image, 1, sizeof(int) * WIDTH * HEIGHT * BYTESPERPIXEL);

	hipStream_t stream;
	hipStreamCreate(&stream);

	//Initialize the data that will reside in the constant gpu memory in the
	float3 lowerLeftCornerCPU, horizontalCPU, verticalCPU, originCPU;
	hipHostAlloc((void**) &lowerLeftCornerCPU, sizeof(float3), hipHostMallocDefault);
	hipHostAlloc((void**) &horizontalCPU, sizeof(float3), hipHostMallocDefault);
	hipHostAlloc((void**) &verticalCPU, sizeof(float3), hipHostMallocDefault);
	hipHostAlloc((void**) &originCPU, sizeof(float3), hipHostMallocDefault);
	lowerLeftCornerCPU = make_float3(-2.0, -1.0, -1.0);
	horizontalCPU = make_float3(4.0, 0.0, 0.0);
	verticalCPU = make_float3(0.0, 2.0, 0.0);
	originCPU = make_float3(0.0, 0.0, 0.0);

	//Send the data to the constant memory
	hipMemcpyToSymbolAsync(HIP_SYMBOL(lowerLeftCorner), &lowerLeftCornerCPU, sizeof(float3), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(horizontal), &horizontalCPU, sizeof(float3), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(vertical), &verticalCPU, sizeof(float3), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(origin), &originCPU, sizeof(float3), 0, hipMemcpyHostToDevice);
	gpuErrorCheck();

	//Allocate the image memory on the gpu
	int *imageGpu;
	hipMalloc((void**) &imageGpu, sizeof(int) * WIDTH * HEIGHT * BYTESPERPIXEL);
	gpuErrorCheck();

	//Transfer the image to the gpu for the elaboration
	hipMemcpyAsync(imageGpu, image, sizeof(int) * WIDTH * HEIGHT * BYTESPERPIXEL, hipMemcpyHostToDevice, stream);
	gpuErrorCheck();

	sphere *spheresCPU[sn];

	spheresCPU[0] = new sphere(make_float3(0, 0, -1), 0.5, 1);
	spheresCPU[1] = new sphere(make_float3(10, -100.5, -1), 100, 2);

	for (int  i = 0; i <  sn; i++) {
		hipMemcpyToSymbolAsync(HIP_SYMBOL(spheres), spheresCPU[i], sizeof(sphere), sizeof(sphere) * i, hipMemcpyHostToDevice);
	}
	gpuErrorCheck();

	kernel<<<BLKSIZE, THRDSIZE, 0, stream>>>(imageGpu, 2);
	gpuErrorCheck();

	//deallocate the constant data that now reside in the constant memory from the cpu
	//TODO deallocate i float3 che ora sono in memoria costante

	//Take back the image
	hipMemcpyAsync(image, imageGpu,  sizeof(int) * WIDTH * HEIGHT * BYTESPERPIXEL, hipMemcpyDeviceToHost, stream);
	gpuErrorCheck();

	//We have to be sure that all the data are back to the cpu
	hipStreamSynchronize(stream);
	generate(WIDTH, HEIGHT, BYTESPERPIXEL, image);

	hipHostFree(image);
	gpuErrorCheck();

	printf("fine\n");
    return EXIT_SUCCESS;
}
